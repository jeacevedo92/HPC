#include "hip/hip_runtime.h"
#include <cmath>
#include <hip/hip_runtime.h>
#include <cv.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <stdio.h>
#include <time.h>

#define CHANNELS 3
#define MASK_WIDTH 3

__constant__ char M1[MASK_WIDTH * MASK_WIDTH];
__constant__ char M2[MASK_WIDTH * MASK_WIDTH];

using namespace cv;
using namespace std;

__global__ void sobel(unsigned char *in, unsigned char *out, int w, int h) {
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  int Row = blockIdx.y * blockDim.y + threadIdx.y;

  if (Col < w && Row < h) {
    int Gx = 0, Gy = 0;
    int N_start_col = Col - (MASK_WIDTH / 2);
    int N_start_row = Row - (MASK_WIDTH / 2);

    for (int j = 0; j < MASK_WIDTH; j++) {
      for (int k = 0; k < MASK_WIDTH; k++) {
        int curRow = N_start_row + j;
        int curCol = N_start_col + k;

        if (curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
          Gx += in[curRow * w + curCol] * M1[j * MASK_WIDTH + k];
          Gy += in[curRow * w + curCol] * M2[j * MASK_WIDTH + k];
        }
      }
    }
    if (Gx < 0)
      Gx = 0;
    else {
      if (Gx > 255)
        Gx = 255;
    }
    if (Gy < 0)
      Gy = 0;
    else {
      if (Gy > 255)
        Gy = 255;
    }

    out[Row * w + Col] = (unsigned char)sqrtf((Gx * Gx) + (Gy * Gy));
  }
}

int main(int argc, char **argv) {
  // Lectura de la imagen con openCV
  Mat image;
  image = imread("perro.jpg", CV_LOAD_IMAGE_COLOR); // Read the file
  Size s = image.size();
  int width = s.width;
  int height = s.height;

  // Definicion de mascaras
  char h_mask1[] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  char h_mask2[] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

  // Definicion de variables que se manejaran en el device
  unsigned char *d_image_Gray, *h_imageOutput, *out;

  // Reserva de memora para variales en host
  h_imageOutput =
      (unsigned char *)malloc(sizeof(unsigned char) * width * height);

  // Reserva de memoria para variables en device
  hipMalloc((void **)&d_image_Gray, sizeof(unsigned char) * width * height);
  hipMalloc((void **)&out, sizeof(unsigned char) * width * height);

  // Definicion de los bloques e hilos por bloques
  int blockSize = 32;
  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(width / float(blockSize)), ceil(height / float(blockSize)),
               1);

  // Copiando los datos del host al device
  hipMemcpyToSymbol(HIP_SYMBOL(M1), h_mask1, sizeof(char) * MASK_WIDTH * MASK_WIDTH);
  hipMemcpyToSymbol(HIP_SYMBOL(M2), h_mask2, sizeof(char) * MASK_WIDTH * MASK_WIDTH);

  // Convirtiendo imagen en escala de grises con openCV
  Mat grayImg;
  cvtColor(image, grayImg, CV_BGR2GRAY);

  // Copiando la imagen del host al device
  hipMemcpy(d_image_Gray, grayImg.data, width * height * sizeof(unsigned char),
             hipMemcpyHostToDevice);

  // Lanzando el kernel
  sobel<<<dimGrid, dimBlock>>>(d_image_Gray, out, width, height);

  // Copiando el resultado del device al host
  hipMemcpy(h_imageOutput, out, width * height * sizeof(unsigned char),
             hipMemcpyDeviceToHost);

  // Generando la imagen de salida
  Mat grayImgCuda;
  grayImgCuda.create(s.height, s.width, CV_8UC1);
  grayImgCuda.data = h_imageOutput;

  // Filtro de Sobel con openCV
  Mat gray_image_opencv, grad_x, abs_grad_x;
  cvtColor(image, gray_image_opencv, CV_BGR2GRAY);
  Sobel(gray_image_opencv, grad_x, CV_8UC1, 1, 0, 3, 1, 0, BORDER_DEFAULT);
  convertScaleAbs(grad_x, abs_grad_x);

  if (!image.data) // Check for invalid input
  {
    cout << "Could not open or find the image" << endl;
    return -1;
  }

  // Guardando la imagen generada por CUDA
  imwrite("perrogris.png", grayImgCuda);

  // Guardando la imagen generada por openCV
  // imwrite("./outputs/1088331150.png", abs_grad_x);
  cout << "La imagen esta lista." << std::endl;
  return 0;
}
