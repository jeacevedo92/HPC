#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

// Multiplicacion de Mini Matriz - Matriz

__global__ void multMatKernel(double *d_a, double *d_b, double *d_c, int NRA,
                              int NCA, int NCB) {
  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;

  if (row < NRA && col < NCB) {
    double result = 0;
    for (int j = 0; j < NCA; j++) {
      result += d_a[row * NCA + j] * d_b[j * NCB + col];
    }
    d_c[row * NCB + col] = result;
  }
}

void multMatCUDA(double *M_a, double *M_b, double *R_c, int NRA, int NCA,
                 int NCB) {
  float blockSize = 32;
  double *d_a, *d_b, *d_c;

  // Asignacion de memoria en el device
  hipMalloc(&d_a, sizeof(double) * NRA * NCA);
  hipMalloc(&d_b, sizeof(double) * NCA * NCB);
  hipMalloc(&d_c, sizeof(double) * NRA * NCB);

  hipMemcpy(d_a, M_a, NRA * NCA * sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(d_b, M_b, NCA * NCB * sizeof(double), hipMemcpyHostToDevice);

  dim3 dimBlock(blockSize, blockSize, 1);
  dim3 dimGrid(ceil(NCB / blockSize), ceil(NRA / blockSize), 1);

  multMatKernel<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, NRA, NCA, NCB);
  hipMemcpy(R_c, d_c, NRA * NCB * sizeof(double), hipMemcpyDeviceToHost);

  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);
}
