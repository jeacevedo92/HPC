#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define SIZE_thread 1024

__global__ void VectorAdd(int *A, int *B, int *C,int n)
{
	int i = threadIdx.x + blockIdx.x*blockDim.x;
	if(i<n)
		C[i]=A[i]+B[i];
}


int main()
{

	int n = 3000;

	clock_t start = clock();

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	a = (int *)malloc(n*sizeof(int));
	b = (int *)malloc(n*sizeof(int));
	c = (int *)malloc(n*sizeof(int));
	
	hipMalloc(&d_a, n*sizeof(int));
	hipMalloc(&d_b, n*sizeof(int));
	hipMalloc(&d_c, n*sizeof(int));

	for(int i=0;i<n;i++)
	{
		a[i]=i;
		b[i]=i;
		c[i]=0;	
	}

	hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, n*sizeof(int), hipMemcpyHostToDevice);

	
	dim3 dimGrid(ceil(n/float(SIZE_thread)),1,1);
	dim3 dimblock(SIZE_thread,1,1);

	VectorAdd<<<dimGrid,dimblock>>>(d_a, d_b, d_c,n);

	hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<10; i++)
		printf("%d ",c[i]);
		
	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);	
	hipFree(d_b);
	hipFree(d_c);
	
	printf("Tiempo transcurrido: %f \n ",((double)clock() - start) / CLOCKS_PER_SEC);

	return 0;
}




