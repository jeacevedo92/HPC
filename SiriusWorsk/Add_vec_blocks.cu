#include "hip/hip_runtime.h"
#include <stdio.h>
#include <malloc.h>
#include <hip/hip_runtime.h>

#define SIZE 65535

__global__ void VectorAdd(int *a, int *b, int *c)
{
	int i = blockIdx.x;
	if(i<SIZE)
		c[i]=a[i]+b[i];
}


int main()
{

	clock_t start = clock();

	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	
	a = (int *)malloc(SIZE*sizeof(int));
	b = (int *)malloc(SIZE*sizeof(int));
	c = (int *)malloc(SIZE*sizeof(int));
	
	hipMalloc(&d_a, SIZE*sizeof(int));
	hipMalloc(&d_b, SIZE*sizeof(int));
	hipMalloc(&d_c, SIZE*sizeof(int));

	for(int i=0;i<SIZE;i++)
	{
		a[i]=i;
		b[i]=i;
		c[i]=0;	
	}

	hipMemcpy(d_a, a, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_c, c, SIZE*sizeof(int), hipMemcpyHostToDevice);

	
	dim3 dimblock(SIZE,1,1);
	dim3 dimGrid(1,1,1);
	
	VectorAdd<<<dimblock,dimGrid>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, SIZE*sizeof(int), hipMemcpyDeviceToHost);

	for(int i=0;i<10; i++)
		printf("%d ",c[i]);
		
	free(a);
	free(b);
	free(c);
	
	hipFree(d_a);	
	hipFree(d_b);
	hipFree(d_c);
	
	printf("Tiempo transcurrido: %f \n ",((double)clock() - start) / CLOCKS_PER_SEC);

	return 0;
}




