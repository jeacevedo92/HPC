//PROGRAMA QUE SUMA DOS MATRICES EN C++

#include<malloc.h>
#include<hip/hip_runtime.h>


#define Size 1024

__global__ void SumaMatricesCU(int* A,int* B,int* C,int width){
	int col=blockIdx.x*blockDim.x + threadIdx.x;//columnas
	int row=blockIdx.y*blockDim.y + threadIdx.y;//filas

	if((row<width)&&(col<width)){
		C[row*width+col] = A[row*width+col]+B[row*width+col];
	}	
}


void imprimeMatriz(int* A, int width){
	for(int i=0;i<width;i++){
		for(int j=0;j<width;j++){
			printf("%d", A[(i*width)+j]);
		}
		printf("\n");
	}

}

void inicializaMatriz(int* X,int width)
{
	for(int i=0; i < width*width ; i++)
	{ 
		X[i]=1;
	}

}


int main()
{


	hipError_t error = hipSuccess;

	int *h_A,*h_B,*h_C,*d_A,*d_B,*d_C;
	
	int width = 2048;

	int size = width * width * sizeof(int);




	// reserva de memoria para las matrices en el host

	h_A = (int*)malloc(size);	
	h_B = (int*)malloc(size);	
	h_C = (int*)malloc(size);



	// inicializa matrices
	inicializaMatriz(h_A,width);
	inicializaMatriz(h_B,width);


	// reserva de memoria para las matrices en el device

	error = hipMalloc((void**)&d_A,size);
	
	if(error != hipSuccess){
		printf("Error reservando memoria para d_M");
		exit(0);
	}
	
	error = hipMalloc((void**)&d_B,size);

	if(error != hipSuccess){
		printf("Error reservando memoria para d_N");
		exit(0);
	}

	error = hipMalloc((void**)&d_C,size);
	
	if(error != hipSuccess){
		printf("Error reservando memoria para d_P");
		exit(0);
	}

	
	//copiando del host al device

	error = hipMemcpy(d_A,h_A,size,hipMemcpyHostToDevice);//destino d_A y origen A
	
	if(error != hipSuccess){
		printf("Error COPIANDO memoria para d_A");
		exit(0);
	}

	error = hipMemcpy(d_B,h_B,size,hipMemcpyHostToDevice);
	
	if(error != hipSuccess){
		printf("Error COPIANDO memoria para d_B");
		exit(0);
	}
	error = hipMemcpy(d_C,h_C,size,hipMemcpyHostToDevice);
	
	if(error != hipSuccess){
		printf("Error COPIANDO memoria para d_C");
		exit(0);
	}

	//47*63*1024=3032064  esta es la cantidad de hilos que vamos a utilizar para hacer la suma de las matrices
	//porque las matrices tienen una dimensión de 2000*1500=3000000 
	//32*32 = 1024 hilos en cada bloque
	//2000/32=63, 1500/32=47
	
	dim3 dimblock(32,32,1);//dimensión de los bloques(cantidad de hilos que se van a utilizar)
	dim3 dimGrid(ceil(width/32),ceil(width/32),1);//dimensión de la malla (cantidad de bloques que se van a utilizar)
	
	SumaMatricesCU<<<dimGrid,dimblock>>>(d_A,d_B,d_C,width);
	
	hipDeviceSynchronize();//espera que termine la funcion anterior 
	
	error = hipMemcpy(h_C,d_C,size,hipMemcpyDeviceToHost);//copia la operacion relizada en el device al host en el vector C
	
	if(error != hipSuccess){
		printf("Error copiando d to h memoria para d_C");
		exit(0);
	}

	imprimeMatriz(h_C,width);
	
	free(h_A);free(h_B);free(h_C);
	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);	
	
	
	return 0;

}

